/**
 * @file testMatrixConv.cu
 * @brief test program to profile convolution layer kernels
 */
 
#include <stdio.h>
#include <stdlib.h>
#include "./inc/common.cuh"
#include "./inc/convolutionLayerFunc.cuh"
#include <string.h>

#define BIAS 0.2f


int main(int argc, char **argv){
	#if (VERBOSE)
		deviceInfor();
	#endif
	
	int iKernel;
	uint_32 nx=1<<10;
	uint_32 ny=1<<10;
	uint_32 blockx=16;
	uint_32 blocky=16;
	int i,j,tile_size;
	
	if (argc<2){
		fprintf(stderr,"usage: <%s> <iKernel> [optional <blockx>] [optional <blocky>] [optional <nx>] [optional <ny>]\n",argv[0]);
	    fprintf(stderr,"ikernel=0: naiveConvLayer\n");
	    fprintf(stderr,"ikernel=1: naiveUnrolledConvLayer\n");
	    fprintf(stderr,"ikernel=2: sharedConvLayer\n");
	    fprintf(stderr,"ikernel=3: sharedUnrollConvLayer\n");
	    fprintf(stderr,"ikernel=4: approxConvLayer\n");
		exit(1);
	}
	
	iKernel=atoi(argv[1]);
	if (argc>2) blockx=atoi(argv[2]);
	if (argc>3) blocky=atoi(argv[3]);
	if (argc>4) nx=atoi(argv[4]);
	if (argc>5) ny=atoi(argv[5]);
	
	dim3 block (blockx, blocky);
    dim3 grid  ((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    double iStart, iElaps;
    double effBW;
    double iGain = 0.0;
    
    int odata_w=nx-KERNEL_SIZE+1;
    int odata_h=ny-KERNEL_SIZE+1;
    
    //data
    float *hSource, *hDest;
    float *dSource, *dDest;
    float *gpuRes;
    float *hFilter;
    float *dFilter;
    //alloc on host
    hSource=(float *)malloc(nx*ny*sizeof(float));
    CHECK_PTR(hSource);
    hDest=(float *)malloc(odata_w*odata_h*sizeof(float));
    CHECK_PTR(hDest);
    hFilter=(float *)malloc(KERNEL_SIZE*KERNEL_SIZE*sizeof(float));
    CHECK_PTR(hFilter);
    gpuRes=(float *)malloc(odata_h*odata_w*sizeof(float));
    CHECK_PTR(gpuRes);
    //alloc on device
    CHECK_CUDA(hipMalloc( (void**)&dSource, nx*ny*sizeof(float)));	
    CHECK_CUDA(hipMalloc( (void**)&dDest, odata_h*odata_w*sizeof(float)));
    CHECK_CUDA(hipMalloc( (void**)&dFilter, KERNEL_SIZE*KERNEL_SIZE*sizeof(float)));
    //init on host
    for(i=0;i<nx*ny;i++)
    	hSource[i]=randomUint8()/(float)1.145f;
    for(i=0;i<KERNEL_SIZE*KERNEL_SIZE;i++)
    	randomUint8()/(float)13.561f;	
   	
   	
    		
    
    #if (VERBOSE)	
    	if(nx<=32 && ny<=32){
    		fprintf(stderr,"Source:\n");
        	for(i=0;i<nx;i++){
        		for(j=0;j<ny;j++){
        			fprintf(stderr,"%.1f ",hSource[i*nx + j]);	
        		}
        		fprintf(stderr,"\n");
        	}
        	fprintf(stderr,"Kernel:\n");
        	for(i=0;i<KERNEL_SIZE;i++){
        		for(j=0;j<KERNEL_SIZE;j++){
        			fprintf(stderr,"%.1f ",hFilter[i*KERNEL_SIZE + j]);	
        		}
        		fprintf(stderr,"\n");
        	}
        }
    #endif
    	
    //copy on GPU
    CHECK_CUDA(hipMemcpy(dSource, hSource, nx*ny*sizeof(float), hipMemcpyHostToDevice));	
    CHECK_CUDA(hipMemcpy(dFilter, hFilter, KERNEL_SIZE*KERNEL_SIZE*sizeof(float), hipMemcpyHostToDevice));	
    
    
    void (* kernel) (float *, float *, int, int, float *, int, float);
    char *kernelName;
    
	    
    switch(iKernel){
    	/*setup naiveConvLayer*/
    	case 0:
    		#if (VERBOSE)
    			fprintf(stdout,"naiveConvLayer kernel selected\n");
    		#endif
    		kernelName=strdup("naiveConvLayer");
    		kernel=&naiveConvLayer;
    		break;
    	/*setup naiveUnrolledConvLayer*/
    	case 1:
    		#if (VERBOSE)
    			fprintf(stdout,"naiveUnrolledConvLayer kernel selected\n");
    		#endif
    		kernelName=strdup("naiveUnrolledConvLayer");
    		kernel=&naiveUnrolledConvLayer;
    		break;
    	/*setup sharedConvLayer*/
    	case 2:
    		#if (VERBOSE)
    			fprintf(stdout,"sharedConvLayer kernel selected\n");
    		#endif
    		kernelName=strdup("sharedConvLayer");
    		kernel=&sharedConvLayer;
   			tile_size = blockx;
    		block.x = (blockx + KERNEL_SIZE -1);
    		block.y = (blockx + KERNEL_SIZE -1);
    		grid.x = (nx + tile_size - 1) / tile_size;
    		grid.y = (ny + tile_size - 1) / tile_size;
    		break;
    	/*setup sharedUnrollConvLayer*/
    	case 3:
    		#if (VERBOSE)
    			fprintf(stdout,"sharedUnrollConvLayer kernel selected\n");
    		#endif
    		kernelName=strdup("sharedUnrollConvLayer");
    		kernel=&sharedUnrollConvLayer;
   			tile_size = blockx;
    		block.x = (blockx + KERNEL_SIZE -1);
    		block.y = (blockx + KERNEL_SIZE -1);
    		grid.x = (nx + tile_size - 1) / tile_size;
    		grid.y = (ny + tile_size - 1) / tile_size;
    		break;
    	/*setup approxConvLayer*/
    	case 4:
    		#if (VERBOSE)
    			fprintf(stdout,"approxConvLayer kernel selected\n");
    		#endif
    		kernelName=strdup("approxConvLayer");
    		kernel=&approxConvLayer;
   			tile_size = blockx;
    		block.x = (blockx + KERNEL_SIZE -1);
    		block.y = (blockx + KERNEL_SIZE -1);
    		grid.x = (nx + tile_size - 1) / tile_size;
    		grid.y = (ny + tile_size - 1) / tile_size;
    		break;
    	default:
    		#if (VERBOSE)
    		fprintf(stderr,"error in kernel selection\n");
    		#endif
    		exit(1);
    		break;
    }
    
    #if (VERBOSE)
    	fprintf(stdout,"nx=%d, ny=%d, %lu Bytes, grid(%d,%d), block(%d,%d), #threads=%llu\n",nx,ny,
    			(nx*ny*sizeof(float)),grid.x,grid.y,
    			 block.x,block.y,(long long unsigned int)(block.x*block.y*grid.x*grid.y));
    #endif
    
    if (iKernel==2 || iKernel ==3 || iKernel==4 || iKernel==5){
    	//dynamic shared memory kernels
    	iStart = cpuSecond();
    	kernel<<<grid,block,(block.x*block.y*sizeof(float))>>>(dSource, dDest, nx, ny, dFilter, KERNEL_SIZE, BIAS);
    	CHECK_CUDA(hipGetLastError());
		CHECK_CUDA(hipDeviceSynchronize());
   		iElaps = cpuSecond() - iStart;
   	}else{
   		//standard kernels with no shared memory
   		iStart = cpuSecond();
    	kernel<<<grid,block>>>(dSource, dDest, nx, ny, dFilter, KERNEL_SIZE, BIAS);
    	CHECK_CUDA(hipGetLastError());
		CHECK_CUDA(hipDeviceSynchronize());
   		iElaps = cpuSecond() - iStart;
   	}
    
    //get data back from gpu
    CHECK_CUDA(hipMemcpy(gpuRes, dDest, odata_h*odata_w*sizeof(float), hipMemcpyDeviceToHost));
    
    #if (CHECK)
    	//compute result on host
    	iStart = cpuSecond();
    	hostConvLayer(hSource, hDest, nx, ny, hFilter, KERNEL_SIZE, BIAS);
    	iGain = cpuSecond() - iStart;
    	iGain = iGain/iElaps;
    	// check kernel results
    	#if (VERBOSE)
    		if (nx<=32 && ny<=32){
    			fprintf(stderr,"CPU result:\n");
        		for(i=0;i<odata_h;i++){
        			for(j=0;j<odata_w;j++){
        				fprintf(stderr,"%.2f ",hDest[i*odata_w + j]);	
        			}
        			fprintf(stderr,"\n");
        		}
        		fprintf(stderr,"GPU result:\n");
        		for(i=0;i<odata_h;i++){
        			for(j=0;j<odata_w;j++){
        				fprintf(stderr,"%.2f ",gpuRes[i*odata_w + j]);	
        			}
        			fprintf(stderr,"\n");
        		}
    		}
    	#endif
        if(checkRes(hDest,gpuRes,odata_w,odata_h)==1){
        	fprintf(stderr,"GPU and CPU result missmatch!\n");		
        	exit(1);
        }
    #endif

    // calculate effective_bandwidth (MB/s)
    effBW=(2 * nx * ny * sizeof(float)) / ((1e+6f)*iElaps);
    /*printf on stdout used for profiling <kernelName>,<elapsedTime>,<bandwidth>,<gain>,<grid(x,y)>,<block(x,y)>*/
    fprintf(stdout,"%s,%f,%f,%f,grid(%d.%d),block(%d.%d)\n",kernelName, effBW, iElaps, iGain, grid.x, grid.y, block.x, block.y);

    // free host and device memory
    CHECK_CUDA(hipFree(dSource));
    CHECK_CUDA(hipFree(dDest));
	CHECK_CUDA(hipFree(dFilter));
	free(hSource);
    free(hDest);
    free(hFilter);
    free(gpuRes);
    
    // reset device
    CHECK_CUDA(hipDeviceReset());
	
	return 0;
}



