/**
 * @file testMatrixConv.cu
 * @brief test program to profile matrixConvolution kernels
 */
 
#include <stdio.h>
#include <stdlib.h>
#include "./inc/common.h"
#include "./inc/matrix2Dconvolution.h"
#include <string.h>

#define KERNEL_SIZE 5

int main(int argc, char **argv){
	#if (VERBOSE)
		deviceInfor();
	#endif
	
	int iKernel;
	uint_32 nx=1<<12;
	uint_32 ny=1<<12;
	uint_32 blockx=16;
	uint_32 blocky=16;
	uint_32 i,j;
	
	if (argc<2){
		fprintf(stderr,"usage: <%s> <iKernel> [optional <blockx>] [optional <blocky>] [optional <nx>] [optional <ny>]\n",argv[0]);
	    fprintf(stderr,"ikernel=0: naive2Dconvolution\n");
		exit(1);
	}
	
	iKernel=atoi(argv[1]);
	if (argc>2) blockx=atoi(argv[2]);
	if (argc>3) blocky=atoi(argv[3]);
	if (argc>4) nx=atoi(argv[4]);
	if (argc>5) ny=atoi(argv[5]);
	
	dim3 block (blockx, blocky);
    dim3 grid  ((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    double iStart, iElaps;
    double effBW;
    double iGain = 0.0;
    
    //data
    float *hSource, *hDest;
    float *dSource, *dDest;
    float *gpuRes;
    float *hFilter;
    float *dFilter;
    //alloc on host
    hSource=(float *)malloc(nx*ny*sizeof(float));
    CHECK_PTR(hSource);
    hDest=(float *)malloc(nx*ny*sizeof(float));
    CHECK_PTR(hDest);
    hFilter=(float *)malloc(KERNEL_SIZE*KERNEL_SIZE*sizeof(float));
    CHECK_PTR(hFilter);
    gpuRes=(float *)malloc(nx*ny*sizeof(float));
    CHECK_PTR(gpuRes);
    //alloc on device
    CHECK_CUDA(hipMalloc( (void**)&dSource, nx*ny*sizeof(float)));	
    CHECK_CUDA(hipMalloc( (void**)&dDest, nx*ny*sizeof(float)));
    CHECK_CUDA(hipMalloc( (void**)&dFilter, KERNEL_SIZE*KERNEL_SIZE*sizeof(float)));
    //init on host
    for(i=0;i<nx*ny;i++)
    	hSource[i]=randomUint8()/(float)1.145f;
    for(i=0;i<KERNEL_SIZE*KERNEL_SIZE;i++)
    	hFilter[i]=randomUint8()/(float)13.561f;
    //copy on GPU
    CHECK_CUDA(hipMemcpy(dSource, hSource, nx*ny*sizeof(float), hipMemcpyHostToDevice));	
    CHECK_CUDA(hipMemcpy(dFilter, hFilter, KERNEL_SIZE*KERNEL_SIZE*sizeof(float), hipMemcpyHostToDevice));	
    
    #if (VERBOSE)
    fprintf(stdout,"nx=%d, ny=%d, %lu Bytes, grid(%d,%d), block(%d,%d), #threads=%llu\n",nx,ny,
    			(nx*ny*sizeof(float)),grid.x,grid.y,
    			 block.x,block.y,(long long unsigned int)(block.x*block.y*grid.x*grid.y));
    #endif
    
    void (* kernel) (float *, float *, uint_32, uint_32, float *, uint_32);
    char *kernelName;
    
    switch(iKernel){
    	/*setup */
    	case 0:
    		#if (VERBOSE)
    		fprintf(stdout,"naive2Dconvolution kernel selected\n");
    		#endif
    		kernelName=strdup("naive2Dconvolution ");
    		kernel=&naive2Dconvolution;
    		break;
    	
    	default:
    		#if (VERBOSE)
    		fprintf(stderr,"error in kernel selection\n");
    		#endif
    		exit(1);
    		break;
    }
    iStart = cpuSecond();
    kernel<<<grid,block>>>(dSource, dDest, nx, ny, dFilter, KERNEL_SIZE);
    CHECK_CUDA(hipGetLastError());
	CHECK_CUDA(hipDeviceSynchronize());
   	iElaps = cpuSecond() - iStart;
    
    //get data back from gpu
    CHECK_CUDA(hipMemcpy(gpuRes, dDest, nx*ny*sizeof(float), hipMemcpyDeviceToHost));
    
    #if (CHECK)
    	//compute result on host
    	iStart = cpuSecond();
    	host2Dconvolution(hSource, hDest, nx, ny, hFilter, KERNEL_SIZE);
    	iGain = cpuSecond() - iStart;
    	iGain = iGain/iElaps;
    	// check kernel results
    	#if (VERBOSE)
    		if (nx<=32 && ny<=32){
    			fprintf(stderr,"Source:\n");
        		for(i=0;i<nx;i++){
        			for(j=0;j<ny;j++){
        				fprintf(stderr,"%.1f ",hSource[i*nx + j]);	
        			}
        			fprintf(stderr,"\n");
        		}
        		fprintf(stderr,"Kernel:\n");
        		for(i=0;i<KERNEL_SIZE;i++){
        			for(j=0;j<KERNEL_SIZE;j++){
        				fprintf(stderr,"%.1f ",hFilter[i*KERNEL_SIZE + j]);	
        			}
        			fprintf(stderr,"\n");
        		}
    			fprintf(stderr,"CPU result:\n");
        		for(i=0;i<nx;i++){
        			for(j=0;j<ny;j++){
        				fprintf(stderr,"%.1f ",hDest[i*nx + j]);	
        			}
        			fprintf(stderr,"\n");
        		}
        		fprintf(stderr,"GPU result:\n");
        		for(i=0;i<nx;i++){
        			for(j=0;j<ny;j++){
        				fprintf(stderr,"%.1f ",gpuRes[i*nx + j]);	
        			}
        			fprintf(stderr,"\n");
        		}
    		}
    	#endif
        if(checkRes(hDest,gpuRes,nx,ny)==1){
        	fprintf(stderr,"GPU and CPU result missmatch!\n");		
        	exit(1);
        }
    #endif

    // calculate effective_bandwidth (MB/s)
    effBW=(2 * nx * ny * sizeof(float)) / ((1e+6f)*iElaps);
    /*printf on stdout used for profiling <kernelName>,<elapsedTime>,<bandwidth>,<gain>,<grid(x,y)>,<block(x,y)>*/
    fprintf(stdout,"%s,%f,%f,%f,grid(%d.%d),block(%d.%d)\n",kernelName, effBW, iElaps, iGain, grid.x, grid.y, block.x, block.y);

    // free host and device memory
    CHECK_CUDA(hipFree(dSource));
    CHECK_CUDA(hipFree(dDest));
	CHECK_CUDA(hipFree(dFilter));
	free(hSource);
    free(hDest);
    free(hFilter);
    free(gpuRes);
    
    // reset device
    CHECK_CUDA(hipDeviceReset());
	
	return 0;
}



